﻿#include "normals.cuh"
#include "hip/hip_runtime.h"
#include ""
#include <vector>
#include <stdio.h>
#include <hiprand.h>
#include <cmath>
#include <thrust/device_vector.h>
#include <iostream>

static hiprandGenerator_t curandGenerator = nullptr;


static void initRandGen() {
	hiprandCreateGenerator(&curandGenerator, HIPRAND_RNG_PSEUDO_MTGP32);
	hiprandSetPseudoRandomGeneratorSeed(curandGenerator, 1234ULL);
}


[[nodiscard]] std::vector<float> cudaNormals(const int steps) {

	if (curandGenerator == nullptr) initRandGen();

	const float dt = 1.0f / float(steps);

	float* gpuPtr;
	hipMalloc(&gpuPtr, steps * sizeof(float));

	hiprandGenerateNormal(curandGenerator, gpuPtr, steps, 0.0f, 1.0f);
	thrust::device_ptr<float> gpu_vec(gpuPtr);

	std::vector<float> output_vec(steps);

	hipMemcpy(output_vec.data(), gpuPtr, steps * sizeof(float), hipMemcpyDeviceToHost);

	hipFree(gpuPtr);

	return output_vec;
}